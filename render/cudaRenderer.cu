#include "hip/hip_runtime.h"
#include <string>
#include <algorithm>
#include <math.h>
#include <stdio.h>
#include <vector>
#include <iostream>
#include <stdexcept>
#include "cycleTimer.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaRenderer.h"
#include "image.h"
#include "noise.h"
#include "sceneLoader.h"
#include "util.h"

#define BLOCKSIZE 256
#define SCAN_BLOCK_DIM   BLOCKSIZE  // needed by sharedMemExclusiveScan implementation
#include "exclusiveScan.cu_inl"
#include "circleBoxTest.cu_inl"

#define DEBUG

#ifdef DEBUG
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr, "CUDA Error: %s at %s:%d\n",
        hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
#else
#define cudaCheckError(ans) ans
#endif

////////////////////////////////////////////////////////////////////////////////////////
// Putting all the cuda kernels here
///////////////////////////////////////////////////////////////////////////////////////

struct GlobalConstants {

    SceneName sceneName;

    int numCircles;
    float* position;
    float* velocity;
    float* color;
    float* radius;

    int imageWidth;
    int imageHeight;
    float* imageData;

    int numCirclesUp;
    int numGridCells;
    int numRegions;
    int maxRegionsPerSmall;
    float smallSize;
};

// Global variable that is in scope, but read-only, for all cuda
// kernels.  The __constant__ modifier designates this variable will
// be stored in special "constant" memory on the GPU. (we didn't talk
// about this type of memory in class, but constant memory is a fast
// place to put read-only variables).
__constant__ GlobalConstants cuConstRendererParams;

// read-only lookup tables used to quickly compute noise (needed by
// advanceAnimation for the snowflake scene)
__constant__ int    cuConstNoiseYPermutationTable[256];
__constant__ int    cuConstNoiseXPermutationTable[256];
__constant__ float  cuConstNoise1DValueTable[256];

// color ramp table needed for the color ramp lookup shader
#define COLOR_MAP_SIZE 5
__constant__ float  cuConstColorRamp[COLOR_MAP_SIZE][3];

// including parts of the CUDA code from external files to keep this
// file simpler and to seperate code that should not be modified
#include "noiseCuda.cu_inl"
#include "lookupColor.cu_inl"

static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// kernelClearImageSnowflake -- (CUDA device code)
//
// Clear the image, setting the image to the white-gray gradation that
// is used in the snowflake image
__global__ void kernelClearImageSnowflake() {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float shade = .4f + .45f * static_cast<float>(height-imageY) / height;
    float4 value = make_float4(shade, shade, shade, 1.f);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelClearImage --  (CUDA device code)
//
// Clear the image, setting all pixels to the specified color rgba
__global__ void kernelClearImage(float r, float g, float b, float a) {

    int imageX = blockIdx.x * blockDim.x + threadIdx.x;
    int imageY = blockIdx.y * blockDim.y + threadIdx.y;

    int width = cuConstRendererParams.imageWidth;
    int height = cuConstRendererParams.imageHeight;

    if (imageX >= width || imageY >= height)
        return;

    int offset = 4 * (imageY * width + imageX);
    float4 value = make_float4(r, g, b, a);

    // write to global memory: As an optimization, I use a float4
    // store, that results in more efficient code than if I coded this
    // up as four seperate fp32 stores.
    *(float4*)(&cuConstRendererParams.imageData[offset]) = value;
}

// kernelAdvanceFireWorks
// 
// Update the position of the fireworks (if circle is firework)
__global__ void kernelAdvanceFireWorks() {
    const float dt = 1.f / 60.f;
    const float pi = 3.14159;
    const float maxDist = 0.25f;

    float* velocity = cuConstRendererParams.velocity;
    float* position = cuConstRendererParams.position;
    float* radius = cuConstRendererParams.radius;

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles)
        return;

    if (0 <= index && index < NUM_FIREWORKS) { // firework center; no update 
        return;
    }

    // determine the fire-work center/spark indices
    int fIdx = (index - NUM_FIREWORKS) / NUM_SPARKS;
    int sfIdx = (index - NUM_FIREWORKS) % NUM_SPARKS;

    int index3i = 3 * fIdx;
    int sIdx = NUM_FIREWORKS + fIdx * NUM_SPARKS + sfIdx;
    int index3j = 3 * sIdx;

    float cx = position[index3i];
    float cy = position[index3i+1];

    // update position
    position[index3j] += velocity[index3j] * dt;
    position[index3j+1] += velocity[index3j+1] * dt;

    // fire-work sparks
    float sx = position[index3j];
    float sy = position[index3j+1];

    // compute vector from firework-spark
    float cxsx = sx - cx;
    float cysy = sy - cy;

    // compute distance from fire-work 
    float dist = sqrt(cxsx * cxsx + cysy * cysy);
    if (dist > maxDist) { // restore to starting position 
        // random starting position on fire-work's rim
        float angle = (sfIdx * 2 * pi)/NUM_SPARKS;
        float sinA = sin(angle);
        float cosA = cos(angle);
        float x = cosA * radius[fIdx];
        float y = sinA * radius[fIdx];

        position[index3j] = position[index3i] + x;
        position[index3j+1] = position[index3i+1] + y;
        position[index3j+2] = 0.0f;

        // travel scaled unit length 
        velocity[index3j] = cosA/5.0;
        velocity[index3j+1] = sinA/5.0;
        velocity[index3j+2] = 0.0f;
    }
}

// kernelAdvanceHypnosis   
//
// Update the radius/color of the circles
__global__ void kernelAdvanceHypnosis() { 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* radius = cuConstRendererParams.radius; 

    float cutOff = 0.5f;
    // place circle back in center after reaching threshold radisus 
    if (radius[index] > cutOff) { 
        radius[index] = 0.02f; 
    } else { 
        radius[index] += 0.01f; 
    }   
}   


// kernelAdvanceBouncingBalls
// 
// Update the positino of the balls
__global__ void kernelAdvanceBouncingBalls() { 
    const float dt = 1.f / 60.f;
    const float kGravity = -2.8f; // sorry Newton
    const float kDragCoeff = -0.8f;
    const float epsilon = 0.001f;

    int index = blockIdx.x * blockDim.x + threadIdx.x; 
   
    if (index >= cuConstRendererParams.numCircles) 
        return; 

    float* velocity = cuConstRendererParams.velocity; 
    float* position = cuConstRendererParams.position; 

    int index3 = 3 * index;
    // reverse velocity if center position < 0
    float oldVelocity = velocity[index3+1];
    float oldPosition = position[index3+1];

    if (oldVelocity == 0.f && oldPosition == 0.f) { // stop-condition 
        return;
    }

    if (position[index3+1] < 0 && oldVelocity < 0.f) { // bounce ball 
        velocity[index3+1] *= kDragCoeff;
    }

    // update velocity: v = u + at (only along y-axis)
    velocity[index3+1] += kGravity * dt;

    // update positions (only along y-axis)
    position[index3+1] += velocity[index3+1] * dt;

    if (fabsf(velocity[index3+1] - oldVelocity) < epsilon
        && oldPosition < 0.0f
        && fabsf(position[index3+1]-oldPosition) < epsilon) { // stop ball 
        velocity[index3+1] = 0.f;
        position[index3+1] = 0.f;
    }
}

// kernelAdvanceSnowflake -- (CUDA device code)
//
// move the snowflake animation forward one time step.  Updates circle
// positions and velocities.  Note how the position of the snowflake
// is reset if it moves off the left, right, or bottom of the screen.
__global__ void kernelAdvanceSnowflake() {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= cuConstRendererParams.numCircles)
        return;

    const float dt = 1.f / 60.f;
    const float kGravity = -1.8f; // sorry Newton
    const float kDragCoeff = 2.f;

    int index3 = 3 * index;

    float* positionPtr = &cuConstRendererParams.position[index3];
    float* velocityPtr = &cuConstRendererParams.velocity[index3];

    // loads from global memory
    float3 position = *((float3*)positionPtr);
    float3 velocity = *((float3*)velocityPtr);

    // hack to make farther circles move more slowly, giving the
    // illusion of parallax
    float forceScaling = fmin(fmax(1.f - position.z, .1f), 1.f); // clamp

    // add some noise to the motion to make the snow flutter
    float3 noiseInput;
    noiseInput.x = 10.f * position.x;
    noiseInput.y = 10.f * position.y;
    noiseInput.z = 255.f * position.z;
    float2 noiseForce = cudaVec2CellNoise(noiseInput, index);
    noiseForce.x *= 7.5f;
    noiseForce.y *= 5.f;

    // drag
    float2 dragForce;
    dragForce.x = -1.f * kDragCoeff * velocity.x;
    dragForce.y = -1.f * kDragCoeff * velocity.y;

    // update positions
    position.x += velocity.x * dt;
    position.y += velocity.y * dt;

    // update velocities
    velocity.x += forceScaling * (noiseForce.x + dragForce.y) * dt;
    velocity.y += forceScaling * (kGravity + noiseForce.y + dragForce.y) * dt;

    float radius = cuConstRendererParams.radius[index];

    // if the snowflake has moved off the left, right or bottom of
    // the screen, place it back at the top and give it a
    // pseudorandom x position and velocity.
    if ( (position.y + radius < 0.f) ||
         (position.x + radius) < -0.f ||
         (position.x - radius) > 1.f)
    {
        noiseInput.x = 255.f * position.x;
        noiseInput.y = 255.f * position.y;
        noiseInput.z = 255.f * position.z;
        noiseForce = cudaVec2CellNoise(noiseInput, index);

        position.x = .5f + .5f * noiseForce.x;
        position.y = 1.35f + radius;

        // restart from 0 vertical velocity.  Choose a
        // pseudo-random horizontal velocity.
        velocity.x = 2.f * noiseForce.y;
        velocity.y = 0.f;
    }

    // store updated positions and velocities to global memory
    *((float3*)positionPtr) = position;
    *((float3*)velocityPtr) = velocity;
}

// shadePixel -- (CUDA device code)
//
// given a pixel and a circle, determines the contribution to the
// pixel from the circle.  Update of the image is done in this
// function.  Called by kernelRenderCircles()
__device__ __inline__ float4
shadePixel(int circleIndex, float2 pixelCenter, float4 existingColor) {
    float px = cuConstRendererParams.position[circleIndex*3];
    float py = cuConstRendererParams.position[circleIndex*3+1];
    float pz = cuConstRendererParams.position[circleIndex*3+2];

    float diffX = px - pixelCenter.x;
    float diffY = py - pixelCenter.y;
    float pixelDist = diffX * diffX + diffY * diffY;

    float rad = cuConstRendererParams.radius[circleIndex];
    float maxDist = rad * rad;

    // circle does not contribute to the image
    if (pixelDist > maxDist)
        return;

    float3 rgb;
    float alpha;

    // there is a non-zero contribution.  Now compute the shading value

    // suggestion: This conditional is in the inner loop.  Although it
    // will evaluate the same for all threads, there is overhead in
    // setting up the lane masks etc to implement the conditional.  It
    // would be wise to perform this logic outside of the loop next in
    // kernelRenderCircles.  (If feeling good about yourself, you
    // could use some specialized template magic).
    if (cuConstRendererParams.sceneName == SNOWFLAKES || cuConstRendererParams.sceneName == SNOWFLAKES_SINGLE_FRAME) {

        const float kCircleMaxAlpha = .5f;
        const float falloffScale = 4.f;

        float normPixelDist = sqrt(pixelDist) / rad;
        rgb = lookupColor(normPixelDist);

        float maxAlpha = .6f + .4f * (1.f-pz);
        maxAlpha = kCircleMaxAlpha * fmaxf(fminf(maxAlpha, 1.f), 0.f); // kCircleMaxAlpha * clamped value
        alpha = maxAlpha * exp(-1.f * falloffScale * normPixelDist * normPixelDist);

    } else {
        // simple: each circle has an assigned color
        int index3 = 3 * circleIndex;
        rgb = *(float3*)&(cuConstRendererParams.color[index3]);
        alpha = .5f;
    }

    float oneMinusAlpha = 1.f - alpha;

    // BEGIN SHOULD-BE-ATOMIC REGION
    // global memory read

    // float4 existingColor = *imagePtr;
    float4 newColor;
    newColor.x = alpha * rgb.x + oneMinusAlpha * existingColor.x;
    newColor.y = alpha * rgb.y + oneMinusAlpha * existingColor.y;
    newColor.z = alpha * rgb.z + oneMinusAlpha * existingColor.z;
    newColor.w = alpha + existingColor.w;
    return newColor;
    // global memory write
    // *imagePtr = newColor;

    // END SHOULD-BE-ATOMIC REGION
}


// kernelRenderCircles -- (CUDA device code)
//
// Each thread renders a circle.  Since there is no protection to
// ensure order of update or mutual exclusion on the output image, the
// resulting image will be incorrect.
// __global__ void kernelRenderCircles() {

//     int index = blockIdx.x * blockDim.x + threadIdx.x;

//     if (index >= cuConstRendererParams.numCircles)
//         return;

//     int index3 = 3 * index;

//     // read position and radius
//     float3 p = *(float3*)(&cuConstRendererParams.position[index3]);
//     float  rad = cuConstRendererParams.radius[index];

//     // compute the bounding box of the circle. The bound is in integer
//     // screen coordinates, so it's clamped to the edges of the screen.
//     short imageWidth = cuConstRendererParams.imageWidth;
//     short imageHeight = cuConstRendererParams.imageHeight;
//     short minX = static_cast<short>(imageWidth * (p.x - rad));
//     short maxX = static_cast<short>(imageWidth * (p.x + rad)) + 1;
//     short minY = static_cast<short>(imageHeight * (p.y - rad));
//     short maxY = static_cast<short>(imageHeight * (p.y + rad)) + 1;

//     // a bunch of clamps.  Is there a CUDA built-in for this?
//     short screenMinX = (minX > 0) ? ((minX < imageWidth) ? minX : imageWidth) : 0;
//     short screenMaxX = (maxX > 0) ? ((maxX < imageWidth) ? maxX : imageWidth) : 0;
//     short screenMinY = (minY > 0) ? ((minY < imageHeight) ? minY : imageHeight) : 0;
//     short screenMaxY = (maxY > 0) ? ((maxY < imageHeight) ? maxY : imageHeight) : 0;

//     float invWidth = 1.f / imageWidth;
//     float invHeight = 1.f / imageHeight;

//     // for all pixels in the bonding box
//     for (int pixelY=screenMinY; pixelY<screenMaxY; pixelY++) {
//         float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixelY * imageWidth + screenMinX)]);
//         for (int pixelX=screenMinX; pixelX<screenMaxX; pixelX++) {
//             float2 pixelCenterNorm = make_float2(invWidth * (static_cast<float>(pixelX) + 0.5f),
//                                                  invHeight * (static_cast<float>(pixelY) + 0.5f));
//             shadePixel(index, pixelCenterNorm, p, imgPtr);
//             imgPtr++;
//         }
//     }
// }

////////////////////////////////////////////////////////////////////////////////////////


CudaRenderer::CudaRenderer() {
    image = NULL;

    numCircles = 0;
    position = NULL;
    velocity = NULL;
    color = NULL;
    radius = NULL;

    cudaDevicePosition = NULL;
    cudaDeviceVelocity = NULL;
    cudaDeviceColor = NULL;
    cudaDeviceRadius = NULL;
    cudaDeviceImageData = NULL;
}

CudaRenderer::~CudaRenderer() {

    if (image) {
        delete image;
    }

    if (position) {
        delete [] position;
        delete [] velocity;
        delete [] color;
        delete [] radius;
    }

    if (cudaDevicePosition) {
        hipFree(cudaDevicePosition);
        hipFree(cudaDeviceVelocity);
        hipFree(cudaDeviceColor);
        hipFree(cudaDeviceRadius);
        hipFree(cudaDeviceImageData);
    }
}

const Image*
CudaRenderer::getImage() {

    // need to copy contents of the rendered image from device memory
    // before we expose the Image object to the caller

    printf("Copying image data from device\n");

    hipMemcpy(image->data,
               cudaDeviceImageData,
               sizeof(float) * 4 * image->width * image->height,
               hipMemcpyDeviceToHost);

    return image;
}

void
CudaRenderer::loadScene(SceneName scene) {
    sceneName = scene;
    loadCircleScene(sceneName, numCircles, position, velocity, color, radius);
}

void
CudaRenderer::setup() {

    int deviceCount = 0;
    std::string name;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Initializing CUDA for CudaRenderer\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        name = deviceProps.name;

        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n", static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n");
    
    // By this time the scene should be loaded.  Now copy all the key
    // data structures into device memory so they are accessible to
    // CUDA kernels
    //
    // See the CUDA Programmer's Guide for descriptions of
    // hipMalloc and hipMemcpy

    hipMalloc(&cudaDevicePosition, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceVelocity, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceColor, sizeof(float) * 3 * numCircles);
    hipMalloc(&cudaDeviceRadius, sizeof(float) * numCircles);
    hipMalloc(&cudaDeviceImageData, sizeof(float) * 4 * image->width * image->height);

    hipMemcpy(cudaDevicePosition, position, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceVelocity, velocity, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceColor, color, sizeof(float) * 3 * numCircles, hipMemcpyHostToDevice);
    hipMemcpy(cudaDeviceRadius, radius, sizeof(float) * numCircles, hipMemcpyHostToDevice);

    numCirclesUp = nextPow2(numCircles+1);
    numGridCells = 16;
    numRegions = numGridCells * numGridCells;
    maxRegionsPerSmall = 5;
    smallSize = (1/(float) numGridCells) * ((float) maxRegionsPerSmall-1)/2.0;

    imageWidth = image->width;
    imageHeight = image->height;

    // Initialize parameters in constant memory.  We didn't talk about
    // constant memory in class, but the use of read-only constant
    // memory here is an optimization over just sticking these values
    // in device global memory.  NVIDIA GPUs have a few special tricks
    // for optimizing access to constant memory.  Using global memory
    // here would have worked just as well.  See the Programmer's
    // Guide for more information about constant memory.

    GlobalConstants params;
    params.sceneName = sceneName;
    params.numCircles = numCircles;
    params.imageWidth = imageWidth;
    params.imageHeight = imageHeight;
    params.position = cudaDevicePosition;
    params.velocity = cudaDeviceVelocity;
    params.color = cudaDeviceColor;
    params.radius = cudaDeviceRadius;
    params.imageData = cudaDeviceImageData;

    params.numCirclesUp = numCirclesUp;
    params.numGridCells = numGridCells;
    params.numRegions = numRegions;
    params.maxRegionsPerSmall = maxRegionsPerSmall;
    params.smallSize = smallSize;

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstRendererParams), &params, sizeof(GlobalConstants));

    // also need to copy over the noise lookup tables, so we can
    // implement noise on the GPU
    int* permX;
    int* permY;
    float* value1D;
    getNoiseTables(&permX, &permY, &value1D);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseXPermutationTable), permX, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoiseYPermutationTable), permY, sizeof(int) * 256);
    hipMemcpyToSymbol(HIP_SYMBOL(cuConstNoise1DValueTable), value1D, sizeof(float) * 256);

    // last, copy over the color table that's used by the shading
    // function for circles in the snowflake demo

    float lookupTable[COLOR_MAP_SIZE][3] = {
        {1.f, 1.f, 1.f},
        {1.f, 1.f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, .9f, 1.f},
        {.8f, 0.8f, 1.f},
    };

    hipMemcpyToSymbol(HIP_SYMBOL(cuConstColorRamp), lookupTable, sizeof(float) * 3 * COLOR_MAP_SIZE);

}

// allocOutputImage --
//
// Allocate buffer the renderer will render into.  Check status of
// image first to avoid memory leak.
void
CudaRenderer::allocOutputImage(int width, int height) {

    if (image)
        delete image;
    image = new Image(width, height);
}

// clearImage --
//
// Clear's the renderer's target image.  The state of the image after
// the clear depends on the scene being rendered.
void
CudaRenderer::clearImage() {

    // 256 threads per block is a healthy number
    dim3 blockDim(16, 16, 1);
    dim3 gridDim(
        (image->width + blockDim.x - 1) / blockDim.x,
        (image->height + blockDim.y - 1) / blockDim.y);

    if (sceneName == SNOWFLAKES || sceneName == SNOWFLAKES_SINGLE_FRAME) {
        kernelClearImageSnowflake<<<gridDim, blockDim>>>();
    } else {
        kernelClearImage<<<gridDim, blockDim>>>(1.f, 1.f, 1.f, 1.f);
    }
    hipDeviceSynchronize();
}

// advanceAnimation --
//
// Advance the simulation one time step.  Updates all circle positions
// and velocities
void
CudaRenderer::advanceAnimation() {
     // 256 threads per block is a healthy number
    dim3 blockDim(256, 1);
    dim3 gridDim((numCircles + blockDim.x - 1) / blockDim.x);

    // only the snowflake scene has animation
    if (sceneName == SNOWFLAKES) {
        kernelAdvanceSnowflake<<<gridDim, blockDim>>>();
    } else if (sceneName == BOUNCING_BALLS) {
        kernelAdvanceBouncingBalls<<<gridDim, blockDim>>>();
    } else if (sceneName == HYPNOSIS) {
        kernelAdvanceHypnosis<<<gridDim, blockDim>>>();
    } else if (sceneName == FIREWORKS) { 
        kernelAdvanceFireWorks<<<gridDim, blockDim>>>(); 
    }
    hipDeviceSynchronize();
}

__global__ void
kernelRecordSpotsOfCircles(int* regions_to_circles_binary) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numCirclesUp * cuConstRendererParams.numRegions) return;

    //Threads 0..numCircles-1 correspond with region 0
    int this_circle = index % cuConstRendererParams.numCirclesUp;
    int this_region = index / cuConstRendererParams.numCirclesUp;
    if (this_circle >= cuConstRendererParams.numCircles) return;
    
    //Get pos for this circle
    float x = cuConstRendererParams.position[this_circle*3];
    float y = cuConstRendererParams.position[this_circle*3+1];
    float r = cuConstRendererParams.radius[this_circle];

    //Get bounds for this region
    int x_units = this_region % cuConstRendererParams.numGridCells;
    int y_units = this_region / cuConstRendererParams.numGridCells;
    float cell_size = 1/((float) cuConstRendererParams.numGridCells);

    float x_left = x_units * cell_size;
    float x_right = (x_units+1) * cell_size;
    float y_bottom = y_units * cell_size;
    float y_top = (y_units+1) * cell_size;
    int indVal = 1;
    // if (this_circle == indVal) printf("%f %f %f %f\n", x_left, x_right, y_top, y_bottom);
    //if (this_circle == indVal) printf("%f %f %f\n", x, y, r);

    //Check if this region contains this circle

    if (!circleInBoxConservative(x, y, r, x_left, x_right, y_top, y_bottom)) return;
    if (!circleInBox(x, y, r, x_left, x_right, y_top, y_bottom)) return;

    regions_to_circles_binary[index] = 1;
}

__global__ void
exclusive_scan_kernel_up(int N, int* result, int two_d) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int two_dplus1 = 2*two_d;
    int first_index = index*two_dplus1 + two_d - 1;
    int second_index = index*two_dplus1 + two_dplus1 - 1;
    result[second_index] += result[first_index];
}
__global__ void
exclusive_scan_kernel_down(int N, int* result, int two_d) {
    //Use up to add the first num to the second num (part 1)
    //Then use this to set the first num to old value of second num (which is second-first)
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int two_dplus1 = 2*two_d;
    int first_index = index*two_dplus1 + two_d - 1;
    int second_index = index*two_dplus1 + two_dplus1 - 1;
    int temp = result[second_index];
    result[second_index] += result[first_index];
    result[first_index] = temp;
    
}

__global__ void
set_last_to_zero_kernel(int M, int N, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= M) return;
    result[N - 1 + index*N] = 0;
}

void exclusive_scan(int* result, int numCirclesUp, int numRegions) {
    //result is a numregions x pow2numcircles array
    //Exclusive scan each region
    int N = numCirclesUp;
    //upsweep phase
    for (int two_d = 1; two_d <= N/2; two_d*=2) {
        int two_dplus1 = 2*two_d;
        int blocks = (N/two_dplus1 + BLOCKSIZE - 1) / BLOCKSIZE;
        //Bulk task launch for the parallel_for
        for (int region = 0; region < numRegions; region++) {
            exclusive_scan_kernel_up<<<blocks, BLOCKSIZE>>>(N/two_dplus1, result+N*region, two_d);
        }
        cudaCheckError(hipDeviceSynchronize());
    }

    int zero_blocks = (numRegions + BLOCKSIZE - 1) / BLOCKSIZE;
    set_last_to_zero_kernel<<<zero_blocks, BLOCKSIZE>>>(numRegions, N, result);

    //downsweep
    for (int two_d = N/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2*two_d;
        int blocks = (N/two_dplus1 + BLOCKSIZE - 1) / BLOCKSIZE;
        //Bulk task launch for the parallel_for
        for (int region = 0; region < numRegions; region++) {
            exclusive_scan_kernel_down<<<blocks, BLOCKSIZE>>>(N/two_dplus1, result+N*region, two_d);
        }
        cudaCheckError(hipDeviceSynchronize());
    }

}

__global__ void
populate_indices_kernel(int* binary, int* cumulative, int* result) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numRegions*cuConstRendererParams.numCirclesUp) return;
    if (!binary[index]) return;
    int this_region = index / cuConstRendererParams.numCirclesUp;
    int this_circle = index % cuConstRendererParams.numCirclesUp;
    result[this_region * cuConstRendererParams.numCirclesUp + cumulative[index+1] - 1] = this_circle;
}

__global__ void
populate_counts_kernel(int* cumulative, int* counts) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= cuConstRendererParams.numRegions) return;
    counts[index] = cumulative[cuConstRendererParams.numCirclesUp * (index+1) - 1];
}

__global__ void
render_pixel_kernel(bool useData, int* regionTable, int* circlesPerRegion) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    int num_pixels = cuConstRendererParams.imageWidth * cuConstRendererParams.imageHeight;
    if (index >= num_pixels) return;
    int pixel_x = index % cuConstRendererParams.imageWidth;
    int pixel_y = index / cuConstRendererParams.imageHeight;
    float pixelWidth = 1.f / cuConstRendererParams.imageWidth;
    float pixelHeight = 1.f / cuConstRendererParams.imageHeight;
    float pixelCenter_x = pixelWidth * (static_cast<float>(pixel_x) + 0.5f);
    float pixelCenter_y = pixelHeight * (static_cast<float>(pixel_y) + 0.5f);
    float2 pixelCenterNorm = make_float2(pixelCenter_x, pixelCenter_y);
    float4* imgPtr = (float4*)(&cuConstRendererParams.imageData[4 * (pixel_y * cuConstRendererParams.imageWidth + pixel_x)]);
    float4 currentColor = *imgPtr;
    
    //Render all circles
    if (!useData) {
        for (int i = 0; i < cuConstRendererParams.numCircles; i++) {
            currentColor = shadePixel(i, pixelCenterNorm, currentColor);
        }
        *imgPtr = currentColor;
        return;
    }


    float regionWidth = 1.f / cuConstRendererParams.numGridCells;
    float regionHeight = 1.f / cuConstRendererParams.numGridCells;
    int region_x = static_cast<int>(pixelCenter_x / regionWidth);
    int region_y = static_cast<int>(pixelCenter_y / regionHeight);
    int region = region_x + region_y * cuConstRendererParams.numGridCells;

    for (int i = 0; i < circlesPerRegion[region]; i++) {
        int index = regionTable[region*cuConstRendererParams.numCirclesUp + i];
        currentColor = shadePixel(index, pixelCenterNorm, currentColor);
    }
    *imgPtr = currentColor;
}

// rgb, rgby, rand10k, rand100k, rand1M, biglittle, littlebig, pattern, micro2M,
                    //   bouncingballs, fireworks, hypnosis, snow, snowsingle

void
CudaRenderer::render() {

    if (numCircles <= 1218) {
        dim3 blockDim(256, 1);
        dim3 pixelsDim((imageWidth * imageHeight + blockDim.x - 1) / blockDim.x);
        render_pixel_kernel<<<pixelsDim, blockDim>>>(false, nullptr, nullptr);
        cudaCheckError(hipDeviceSynchronize());
        return;
    }

    double startTime = CycleTimer::currentSeconds();

    int length;
    int* vals;
    int* cudaDeviceRegionTableBinary = nullptr;
    int* cudaDeviceRegionTableCumulative = nullptr;
    int* cudaDeviceRegionTable = nullptr;
    int* cudaDeviceCirclesPerRegion = nullptr;
    //binary is numRegions x numCircles
    //cumulative is numRegions x roundup(numCircles+1)
    //table is numRegions x numCircles
    //perR is numRegions x 1
    hipMalloc(&cudaDeviceRegionTableBinary, sizeof(int) * numCirclesUp * numRegions);
    hipMalloc(&cudaDeviceRegionTableCumulative, sizeof(int) * numCirclesUp * numRegions);
    hipMalloc(&cudaDeviceRegionTable, sizeof(int) * numCirclesUp * numRegions);
    hipMalloc(&cudaDeviceCirclesPerRegion, sizeof(int) * numRegions);

    double endTime = CycleTimer::currentSeconds();
    printf("Alloc arrays: %.3f ms\n", 1000.f * (endTime-startTime));


    startTime = CycleTimer::currentSeconds();
    //Now, do a task launch of the kernel over all circles
    dim3 blockDim(256, 1);
    dim3 gridDim((numCirclesUp * numRegions + blockDim.x - 1) / blockDim.x);
    kernelRecordSpotsOfCircles<<<gridDim, blockDim>>>(cudaDeviceRegionTableBinary);
    cudaCheckError(hipDeviceSynchronize());
    endTime = CycleTimer::currentSeconds();
    printf("Alloc arrays: %.3f ms\n", 1000.f * (endTime-startTime));

    // length = numCirclesUp * numRegions;
    // vals = (int*) malloc(length*sizeof(int));
    // hipMemcpy(vals, cudaDeviceRegionTableBinary, length*sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << "Binary" << std::endl;
    // for (int i = 0; i < numRegions; i++) {
    //     for (int j = 0; j < numCirclesUp; j++) {
    //         std::cout << vals[i*numCirclesUp + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;
    
    // hipFree(vals);

    // cudaCheckError(hipDeviceSynchronize());

    //Copy the table and run cumsum on it
    startTime = CycleTimer::currentSeconds();
    hipMemcpy(cudaDeviceRegionTableCumulative, cudaDeviceRegionTableBinary, numCirclesUp * numRegions * sizeof(int), hipMemcpyDeviceToDevice);
    endTime = CycleTimer::currentSeconds();
    printf("Copy table: %.3f ms\n", 1000.f * (endTime-startTime));
    
    startTime = CycleTimer::currentSeconds();
    exclusive_scan(cudaDeviceRegionTableCumulative, numCirclesUp, numRegions);
    endTime = CycleTimer::currentSeconds();
    printf("Exclusive scan: %.3f ms\n", 1000.f * (endTime-startTime));

    // length = numCirclesUp * numRegions;
    // vals = (int*) malloc(length*sizeof(int));
    // hipMemcpy(vals, cudaDeviceRegionTableCumulative, length*sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << "Cumulative" << std::endl;
    // for (int i = 0; i < numRegions; i++) {
    //     for (int j = 0; j < numCirclesUp; j++) {
    //         std::cout << vals[i*numCirclesUp + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;
    
    // hipFree(vals);

    //Generate condensed regions->circles map
    startTime = CycleTimer::currentSeconds();
    populate_indices_kernel<<<gridDim, blockDim>>>(cudaDeviceRegionTableBinary, cudaDeviceRegionTableCumulative, cudaDeviceRegionTable);
    dim3 gridRegionsDim((numRegions + blockDim.x - 1) / blockDim.x);
    populate_counts_kernel<<<gridRegionsDim, blockDim>>>(cudaDeviceRegionTableCumulative, cudaDeviceCirclesPerRegion);
    cudaCheckError(hipDeviceSynchronize());
    endTime = CycleTimer::currentSeconds();
    printf("Populate indices: %.3f ms\n", 1000.f * (endTime-startTime));

    // length = numCirclesUp * numRegions;
    // vals = (int*) malloc(length*sizeof(int));
    // hipMemcpy(vals, cudaDeviceRegionTable, length*sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << "Circles in each region" << std::endl;
    // for (int i = 0; i < numRegions; i++) {
    //     for (int j = 0; j < numCirclesUp; j++) {
    //         std::cout << vals[i*numCirclesUp + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    
    // hipFree(vals);

    //Render everything
    startTime = CycleTimer::currentSeconds();
    dim3 pixelsDim((imageWidth * imageHeight + blockDim.x - 1) / blockDim.x);
    render_pixel_kernel<<<pixelsDim, blockDim>>>(true, cudaDeviceRegionTable, cudaDeviceCirclesPerRegion);
    cudaCheckError(hipDeviceSynchronize());
    endTime = CycleTimer::currentSeconds();
    printf("Rendering pixels: %.3f ms\n", 1000.f * (endTime-startTime));

    hipFree(cudaDeviceRegionTableBinary);
    hipFree(cudaDeviceRegionTableCumulative);
    hipFree(cudaDeviceRegionTable);
    hipFree(cudaDeviceCirclesPerRegion);
}
